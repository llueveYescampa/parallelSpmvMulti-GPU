#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include "real.h"
#include "dataDef.h"

#include "parallelSpmv.h"

#define MAXTHREADS 128
#define REP 1000

#ifdef DOUBLE
    texture<int2>  xTex;
    texture<int2>  valTex;
#else
    texture<float> xTex;
    texture<float> valTex;
#endif

void meanAndSd(real *mean, real *sd,real *data, int n)
{
    real sum = (real) 0.0; 
    real standardDeviation = (real) 0.0;

    for(int i=0; i<n; ++i) {
        sum += data[i];
    } // end for //

    *mean = sum/n;

    for(int i=0; i<n; ++i) {
        standardDeviation += pow(data[i] - *mean, 2);
    } // end for //
    *sd=sqrt(standardDeviation/n);
} // end of calculateSD //


int main(int argc, char *argv[]) 
{
    #include "parallelSpmvData.h"

    hipError_t cuda_ret;
    
    cuda_ret = hipGetDeviceCount(&ngpus);
    if(cuda_ret != hipSuccess) FATAL("Unable to deternine number of GPUs");
    //ngpus=4;

    // verifing number of input parameters //
   char exists='t';
   char checkSol='f';
   
    if (argc < 3 ) {
        printf("Use: %s  Matrix_filename InputVector_filename  [SolutionVector_filename]  \n", argv[0]);     
        exists='f';
    } // endif //
    
    FILE *fh=NULL;
    // testing if matrix file exists
    if((fh = fopen(argv[1], "rb")  )   == NULL) {
        printf("No matrix file found.\n");
        exists='f';
    } // end if //
    
    // testing if input file exists
    if((fh = fopen(argv[2], "rb")  )   == NULL) {
        printf("No input vector file found.\n");
        exists='f';
    } // end if //

    // testing if output file exists
    if (argc  >3 ) {
        if((fh = fopen(argv[3], "rb")  )   == NULL) {
            printf("No output vector file found.\n");
            exists='f';
        } else {
            checkSol='t';
        } // end if //
    } // end if //
    if (fh) fclose(fh);
        
    if (exists == 'f') {
        printf("Quitting.....\n");
        exit(0);
    } // end if //

    printf("%s Precision. Solving using %d %s\n", (sizeof(real) == sizeof(double)) ? "Double": "Single", ngpus, (ngpus > 1) ? "GPUs": "GPU"  );

    #include "parallelSpmvAllocateData.h" 
    
    reader(&n, 
           on_proc_nnz,
           off_proc_nnz,
           &row_ptr,&col_idx,&val,
           &row_ptr_off,&col_idx_off,&val_off,
           argv[1], ngpus);
           
           
    if (ngpus>1) {
        createCommunicator(nColsOff, sendColumns, recvSendCount , col_idx_off, off_proc_nnz, n,&ngpus);
    } // end if //    

    
    // ready to start //    
    
    int totalNNZ=0;
    for (int gpu=0; gpu<ngpus; ++gpu) {
        totalNNZ+=on_proc_nnz[gpu];
        totalNNZ+=off_proc_nnz[gpu];
        
        cuda_ret = hipSetDevice(gpu);
        if(cuda_ret != hipSuccess) FATAL("Unable to set gpu");
    
        //cuda_ret = hipStreamCreateWithFlags(&stream0[gpu], hipStreamDefault);
        cuda_ret = hipStreamCreateWithFlags(&stream[gpu], hipStreamNonBlocking ) ;
        if(cuda_ret != hipSuccess) FATAL("Unable to create stream0 ");
        
        v[gpu] = (real *) malloc((n[gpu])*sizeof(real));
        w[gpu] = (real *) malloc((n[gpu])*sizeof(real));
        vectorReader(v[gpu], &gpu, n, argv[2]);
        if (ngpus > 1) hipHostAlloc((real **)&v_off[gpu]  , nColsOff[gpu]*sizeof(real),hipHostMallocDefault);


        /////////////////////////////////////////////////////
        // determining the standard deviation of the nnz per row
        real *temp=(real *) malloc((n[gpu])*sizeof(real));
        for (int row=0; row<n[gpu]; ++row) {
            temp[row] = row_ptr[gpu][row+1] - row_ptr[gpu][row];
        } // end for //
        meanAndSd(&meanNnzPerRow0[gpu],&sd0[gpu],temp,n[gpu]);
//printf("file: %s, line: %d, gpu on-prcoc:   %d, mean: %7.3f, sd: %7.3f using: %s \n", __FILE__, __LINE__, gpu , meanNnzPerRow0[gpu], sd0[gpu], (meanNnzPerRow0[gpu] + 0.5*sd0[gpu] < 32) ? "spmv0": "spmv1");
        if (nColsOff[gpu]) {
            for (int row=0; row<n[gpu]; ++row) {
                temp[row] = row_ptr_off[gpu][row+1] - row_ptr_off[gpu][row];
            } // end for //
            meanAndSd(&meanNnzPerRow1[gpu],&sd1[gpu],temp,n[gpu]);
//printf("file: %s, line: %d, gpu off-prcoc:  %d, mean: %7.3f, sd: %7.3f using: %s \n", __FILE__, __LINE__, gpu , meanNnzPerRow1[gpu], sd1[gpu], (meanNnzPerRow1[gpu] + 0.5*sd1[gpu] < 32) ? "spmv0": "spmv1");
        } // end if //        
        free(temp);
        /////////////////////////////////////////////////////
    
        hipSetDevice(gpu);
        //printf("file: %s, line: %d, setting gpu: %d\n", __FILE__, __LINE__,gpu);        

       
        // Allocating device memory for on_proc input matrices 

        cuda_ret = hipMalloc((void **) &row_ptr_d[gpu],  (n[gpu]+1)      * sizeof(int) );
        if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for rows_d");
        
        cuda_ret = hipMalloc((void **) &col_idx_d[gpu], on_proc_nnz[gpu] * sizeof(int));
        if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for cols_d");

        cuda_ret = hipMalloc((void **) &val_d[gpu],     on_proc_nnz[gpu] * sizeof(real));
        if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for vals_d");

        // Copy the input on_proc  matrices from the host memory to the device memory
        
        cuda_ret = hipMemcpy(row_ptr_d[gpu], row_ptr[gpu], (n[gpu]+1)*sizeof(int),hipMemcpyHostToDevice);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device matrix row_ptr_d");

        cuda_ret = hipMemcpy(col_idx_d[gpu], col_idx[gpu], on_proc_nnz[gpu]*sizeof(int),hipMemcpyHostToDevice);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device matrix col_idx_d");

        cuda_ret = hipMemcpy(val_d[gpu], val[gpu],         on_proc_nnz[gpu]*sizeof(real),hipMemcpyHostToDevice);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device matrix val_d");

        // Allocating device memory for inpit and output vectors

        cuda_ret = hipMalloc((void **) &(w_d[gpu]),  n[gpu]*sizeof(real));
        if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for w_d");
        
        cuda_ret = hipMalloc((void **) &(v_d[gpu]),  n[gpu]*sizeof(real));
        if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for v_d");

        // Copy the input vector to device

        cuda_ret = hipMemcpy(v_d[gpu], v[gpu], n[gpu]*sizeof(real),hipMemcpyHostToDevice);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device matrix v_d");



        if (ngpus > 1) {
            // Allocating device memory for off_proc input matrices 
            cuda_ret = hipMalloc((void **) &row_ptr_off_d[gpu],  (n[gpu]+1)*sizeof(int));
            if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for rows_d");

            cuda_ret = hipMalloc((void **) &col_idx_off_d[gpu], off_proc_nnz[gpu] * sizeof(int));
            if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for cols_d");

            cuda_ret = hipMalloc((void **) &val_off_d[gpu],  off_proc_nnz[gpu] *sizeof(real));
            if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for vals_d");


            // Copy the input off_proc  matrices from the host memory to the device memory

            cuda_ret = hipMemcpy(col_idx_off_d[gpu], col_idx_off[gpu], off_proc_nnz[gpu]*sizeof(int),hipMemcpyHostToDevice);
            if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device matrix col_idx_d");

            cuda_ret = hipMemcpy(val_off_d[gpu]   , val_off[gpu],      off_proc_nnz[gpu]*sizeof(real),hipMemcpyHostToDevice);
            if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device matrix val_d");

            cuda_ret = hipMemcpy(row_ptr_off_d[gpu], row_ptr_off[gpu], (n[gpu]+1)*sizeof(int),hipMemcpyHostToDevice);
            if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device matrix row_ptr_d");

            // Allocating device memory for inpit and output vectors
            cuda_ret = hipMalloc((void **) &v_off_d[gpu],  nColsOff[gpu] *sizeof(real));
            if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for v_off_d");

        } // end if //


/////////////////////////////////////////////////////////////////////////

        printf("In GPU: %d\n",gpu);
        if (meanNnzPerRow0[gpu] < warpSize && parameter2Adjust*sd0[gpu] < warpSize) {
        	// these mean use scalar spmv
            if (meanNnzPerRow0[gpu] < (real) 4.5) {
                block0[gpu].x=128;
            } else if (meanNnzPerRow0[gpu]< (real) 14.4) {
                block0[gpu].x=64;
            } else {
                block0[gpu].x=32;
            } // end if //

            grid0[gpu].x = ( (n[gpu] + block0[gpu].x -1) /block0[gpu].x );
            printf("using scalar spmv for on matrix,  blockSize: [%d, %d] %f, %f\n",block0[gpu].x,block0[gpu].y, meanNnzPerRow0[gpu], sd0[gpu]) ;
        } else {
            // these mean use vector spmv
            if (meanNnzPerRow0[gpu] > 10.0*warpSize) {
                block0[gpu].x=2*warpSize;
            }  else {
                block0[gpu].x=warpSize;
            } // end if //
            block0[gpu].y=MAXTHREADS/block0[gpu].x;
            grid0[gpu].x = ( (n[gpu] + block0[gpu].y - 1) / block0[gpu].y ) ;
        	sharedMemorySize0[gpu]=block0[gpu].x*block0[gpu].y*sizeof(real);
            printf("using vector spmv for on matrix,  blockSize: [%d, %d] %f, %f\n",block0[gpu].x,block0[gpu].y, meanNnzPerRow0[gpu], sd0[gpu]) ;
        } // end if // 

        if (ngpus > 1) {
            if (meanNnzPerRow1[gpu] < warpSize  && parameter2Adjust*sd1[gpu] < warpSize) {
            	// these mean use scalar spmv
                if (meanNnzPerRow1[gpu] < (real) 4.5) {
                    block1[gpu].x=128;
                } else if (meanNnzPerRow1[gpu] < (real) 14.4) {
                    block1[gpu].x=64;
                } else {
                    block1[gpu].x=32;
                } // end if //
                grid1[gpu].x = ( (n[gpu] + block1[gpu].x -1) /block1[gpu].x );
                printf("using scalar spmv for off matrix, blockSize: [%d, %d] %f, %f\n",block1[gpu].x,block1[gpu].y, meanNnzPerRow1[gpu], sd1[gpu]) ;
            } else {
                // these mean use vector spmv
                if (meanNnzPerRow1[gpu] > 10.0*warpSize) {
                    block1[gpu].x=2*warpSize;
                }  else {
                    block1[gpu].x=warpSize;
                } // end if //

                block1[gpu].y=MAXTHREADS/block1[gpu].x;
                grid1[gpu].x = ( (n[gpu] + block1[gpu].y - 1) / block1[gpu].y ) ;
            	sharedMemorySize1[gpu]=block1[gpu].x*block1[gpu].y*sizeof(real);
                printf("using vector spmv for off matrix, blockSize: [%d, %d] %f, %f\n",block1[gpu].x,block1[gpu].y, meanNnzPerRow1[gpu], sd1[gpu]) ;
            } // end if // 
        }
    } // end for //

    // Timing should begin here//
    struct timeval tp;                                   // timer
    double elapsed_time;
    
    gettimeofday(&tp,NULL);  // Unix timer
    elapsed_time = -(tp.tv_sec*1.0e6 + tp.tv_usec);
    
    for (int t=0; t<REP; ++t) {
        // send the first spmv
        for (int gpu=0; gpu<ngpus; ++gpu) {
            hipSetDevice(gpu);
        
            cuda_ret = hipMemset(w_d[gpu], 0, sizeof(real)*n[gpu] );
            if(cuda_ret != hipSuccess) FATAL("Unable to set device for matrix w_d[gpu]");

            cuda_ret = hipBindTexture(NULL, xTex,   v_d[gpu],   n[gpu]           * sizeof(real));
            cuda_ret = hipBindTexture(NULL, valTex, val_d[gpu], on_proc_nnz[gpu] * sizeof(real));
            spmv<<<grid0[gpu], block0[gpu], sharedMemorySize0[gpu]>>>(w_d[gpu],  row_ptr_d[gpu], col_idx_d[gpu], n[gpu]);
            cuda_ret = hipUnbindTexture(xTex);
            cuda_ret = hipUnbindTexture(valTex);
            
        } // end for //
        
        if (ngpus > 1 ) {
            startComunication(v,v_off,recvSendCount, sendColumns, &ngpus);
            // send the second spmv
            for (int gpu=0; gpu<ngpus; ++gpu) {
                hipSetDevice(gpu);
            
                cuda_ret = hipMemcpyAsync(v_off_d[gpu], v_off[gpu], nColsOff[gpu]*sizeof(real),hipMemcpyHostToDevice,stream[gpu] ) ;
                if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device array v_off_d");
            
                cuda_ret = hipBindTexture(NULL, xTex,   v_off_d[gpu],   nColsOff[gpu]     * sizeof(real));
                cuda_ret = hipBindTexture(NULL, valTex, val_off_d[gpu], off_proc_nnz[gpu] * sizeof(real));
                spmv<<<grid1[gpu], block1[gpu], sharedMemorySize1[gpu],stream[gpu] >>>(w_d[gpu],  row_ptr_off_d[gpu], col_idx_off_d[gpu], n[gpu]);
                cuda_ret = hipUnbindTexture(xTex);
                cuda_ret = hipUnbindTexture(valTex);
                
            } // end for //
        } // end if //

        for (int gpu=0; gpu<ngpus; ++gpu) {
            hipSetDevice(gpu);
            hipStreamSynchronize(NULL);
            hipStreamSynchronize(stream[gpu]);
        } // end for //
    } // end for //
    
    gettimeofday(&tp,NULL);
    elapsed_time += (tp.tv_sec*1.0e6 + tp.tv_usec);
    printf ("Total time was %f seconds, GFLOPS: %f\n", elapsed_time*1.0e-6,  2.0*totalNNZ*REP*1.0e-3/elapsed_time  );
    
    for (int gpu=0; gpu<ngpus; ++gpu) {
        hipSetDevice(gpu);
        cuda_ret = hipMemcpy(w[gpu], w_d[gpu], n[gpu]*sizeof(real),hipMemcpyDeviceToHost);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device matrix y_d back to host");
    } // end for //




    if (checkSol=='t') {

        for (int gpu=0; gpu<ngpus; ++gpu) {
            real *sol= (real *) malloc( n[gpu] * sizeof(real));  
            hipSetDevice(gpu);
            // reading input vector
            vectorReader(sol, &gpu, n, argv[3]);
            
            int row=0;
            real tolerance = 1.0e-08;
            if (sizeof(real) != sizeof(double) ) {
                tolerance = 1.0e-02;
            } // end if //
            real error;

            do {
                error =  fabs(sol[row] - w[gpu][row]) /fabs(sol[row]);
                if ( error > tolerance ) break;
                ++row;
            } while (row < n[gpu]); // end do-while //
            
            if (row == n[gpu]) {
                printf("Solution match in gpu %d\n",gpu);
            } else {    
                printf("For Matrix %s, solution does not match at element %d in gpu %d   %20.13e   -->  %20.13e  error -> %20.13e, tolerance: %20.13e \n", 
                argv[1], (row+1),gpu, sol[row], w[gpu][row], error , tolerance  );
            } // end if //
            free(sol);    
        } // end for //

        
    } // end if //

    #include "parallelSpmvCleanData.h" 
    return 0;    
//    printf("file: %s, line: %d, so far so good\n", __FILE__, __LINE__ ); exit(0);
} // end main() //
